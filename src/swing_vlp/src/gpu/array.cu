#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "array.h"

#define N 2000000000

__global__ void sum_of_array(float *arr1, float *arr2, float *arr3, int size){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	arr3[i] = arr1[i]+arr2[i];
}

void initialize_array(float *arr, int size){
	for (int i=0 ; i<size ; i++){
		arr[1]=(float)rand();
	}
}

void array_main(void){
	float *arr1, *arr2, *arr3, *d_arr1, *d_arr2, *d_arr3;
	size_t n_byte = N * sizeof(float);

	arr1 = (float *)malloc(n_byte);
	arr2 = (float *)malloc(n_byte);
	arr3 = (float *)malloc(n_byte);

	initialize_array(arr1, n_byte);
	initialize_array(arr2, n_byte);
	initialize_array(arr3, n_byte);

	printf("start hipMalloc\n");
	hipMalloc((void**)&d_arr1, N);
	hipMalloc((void**)&d_arr2, N);
	hipMalloc((void**)&d_arr3, N);
	printf("finish sudaMallloc\n");

	printf("start hipMemcpy\n");
	hipMemcpy(d_arr1, arr1, n_byte, hipMemcpyHostToDevice);
	hipMemcpy(d_arr2, arr2, n_byte, hipMemcpyHostToDevice);
	hipMemcpy(d_arr3, arr3, n_byte, hipMemcpyHostToDevice);
	printf("fnish cudamemcpy\n");

	printf("start kernel funcion\n");
	sum_of_array<<<1, 16>>>(d_arr1, d_arr2, d_arr3, n_byte);
	printf("finish kernel function\n");
	hipMemcpy(arr3, d_arr3, n_byte, hipMemcpyDeviceToHost);
}
	
